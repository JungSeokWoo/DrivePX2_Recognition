#include "hip/hip_runtime.h"
#include "px2camlib.h"

// Convert Cam Img to gpuMat
__global__
void PitchedRGBA2GpuMat(uint8_t* pitchedImgRGBA, uint8_t* imgGpuMat, int width, int height, int cudaPitch)
{
    int xIndex_3ch = blockIdx.x*blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y*blockDim.y + threadIdx.y;

    if((xIndex_3ch < 3*width) && (yIndex < height))
    {
        int xIndex = xIndex_3ch%width;
        int c = xIndex_3ch/width;

        int j = yIndex*width + xIndex;
        imgGpuMat[j*3 + 2 - c] = pitchedImgRGBA[cudaPitch*yIndex + c + xIndex*4];
    }
}

// Crop and convert gpuMat original image to Tensor RT and gpuMat
__global__
void GpuMat2Img(uint8_t* imgGpuMatOri, float* imgTrt, uint8_t* imgGpuMat,
                int width, int height,
                int roiX, int roiY, int roiW, int roiH)
{
    int xIndex_3ch = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if((xIndex_3ch < 3*roiW) && (yIndex < roiH))
    {
        int xIndex = xIndex_3ch%roiW;
        int c = xIndex_3ch/roiW;

        int j = (yIndex)*roiW + (xIndex);
        int j_ori = (yIndex + roiY)*width + (xIndex + roiX);

        imgTrt[c*roiH*roiW + j] = (float)imgGpuMatOri[j_ori*3 + 2 - c]/255.f;
        imgGpuMat[j*3 + 2 - c] = imgGpuMatOri[j_ori*3 + 2 - c];
    }
}

px2Cam::px2Cam()
{
    mArguments = ProgramArguments(
    {           ProgramArguments::Option_t("camera-type", "ar0231-rccb-ae-sf3324"),
                ProgramArguments::Option_t("custom-board", "1"),
                ProgramArguments::Option_t("csi-port", "ab"),
                ProgramArguments::Option_t("write-file", ""),
                ProgramArguments::Option_t("serializer-type", "h264"),
                ProgramArguments::Option_t("serializer-bitrate", "8000000"),
                ProgramArguments::Option_t("serializer-framerate", "30"),
                ProgramArguments::Option_t("slave", "0")
    });
}

px2Cam::~px2Cam()
{
    ReleaseModules();
}

void px2Cam::ReleaseModules()
{
    if(mStreamerCUDA2GL)
    {
        dwImageStreamer_release(&mStreamerCUDA2GL);
    }

    if(mCamera)
    {
        dwSensor_stop(mCamera);
        dwSAL_releaseSensor(&mCamera);
    }

    if(mRenderEngine)
    {
        dwRenderEngine_release(&mRenderEngine);
    }

    if(mRenderer)
    {
        dwRenderer_release(&mRenderer);
    }

    dwSAL_release(&mSAL);
    dwRelease(&mContext);
}

bool px2Cam::Init(imgCropParameters imgCropParams,
                  displayParameters dispParams,
                  dwTegraMode tegraMode,
                  const char* writePath)
{
    mArguments.set("write-file", writePath);
    return Init(imgCropParams, dispParams, tegraMode);
}

bool px2Cam::Init(imgCropParameters imgCropParams,
                  displayParameters dispParams,
                  dwTegraMode tegraMode)
{
    // Set Resize and ROI info

    mRecordCamera = !mArguments.get("write-file").empty();
    mTegraMode = tegraMode;
    mDispParams = dispParams;

    if (abs(imgCropParams.resizeRatio - 1.0) < 0.001)
    {
        mResizeEnable = false;
    }
    else
    {
        mResizeEnable = true;
        mResizeRatio = imgCropParams.resizeRatio;
        mResizeWidth = (int)((float)CAM_IMG_WIDTH*imgCropParams.resizeRatio);
        mResizeHeight = (int)((float)CAM_IMG_HEIGHT*imgCropParams.resizeRatio);
    }

    mROIx = imgCropParams.roiX;
    mROIy = imgCropParams.roiY;
    mROIw = imgCropParams.roiW;
    mROIh = imgCropParams.roiH;

    // Check ROI is inrange of Camera Image
    if(!mResizeEnable)
    {
        int roiBRx = mROIx + mROIw;
        int roiBRy = mROIy + mROIh;

        if((mROIx < 0) && (mROIx > CAM_IMG_WIDTH) ||
                (mROIy < 0) || (mROIy > CAM_IMG_HEIGHT) ||
                (roiBRx < 0) || (roiBRx > CAM_IMG_WIDTH) ||
                (roiBRy < 0) || (roiBRy > CAM_IMG_HEIGHT))
        {
            cout << "ROI is out of range..." << "Camera image resolution : (" << CAM_IMG_WIDTH << "," << CAM_IMG_HEIGHT << ")"
                 << "...But ROI is : " << "(" << mROIx << "," << mROIy << ") ~ (" << roiBRx << "," << roiBRy << ")" <<  endl;
            return false;
        }
    }
    else
    {
        int roiBRx = mROIx + mROIw;
        int roiBRy = mROIy + mROIh;

        int roiTLxOri;
        int roiTLyOri;
        int roiBRxOri;
        int roiBRyOri;

        CoordTrans_Resize2Ori(mROIx, mROIy, roiTLxOri, roiTLyOri);
        CoordTrans_Resize2Ori(roiBRx, roiBRy, roiBRxOri, roiBRyOri);

        if((roiTLxOri < 0) || (roiTLxOri > CAM_IMG_WIDTH) ||
                (roiTLyOri < 0) || (roiTLyOri > CAM_IMG_HEIGHT) ||
                (roiBRxOri < 0) || (roiBRxOri > CAM_IMG_WIDTH) ||
                (roiBRyOri < 0) || (roiBRyOri > CAM_IMG_HEIGHT))
        {
            cout << "ROI is out of range" << "Camera image resolution : (" << CAM_IMG_WIDTH << "," << CAM_IMG_HEIGHT << ")"
                 << "...But ROI is : " << "(" << roiTLxOri << "," << roiTLyOri << ") ~ (" << roiBRxOri << "," << roiBRyOri << ")" <<  endl;
            return false;
        }
    }

    // Initialize Modules
    bool status;
    InitGL();

    status = InitSDK();
    if(!status)
        return status;

    status = InitRenderer();
    if(!status)
        return status;

    status = InitSAL();
    if(!status)
        return status;

    status = InitSensors();
    if(!status)
        return status;

    status = InitPipeline();
    if(!status)
        return status;

    return true;
}

void px2Cam::CoordTrans_Resize2Ori(int xIn, int yIn, int& xOut, int& yOut)
{
    xOut = (int)(xIn/mResizeRatio);
    yOut = (int)(yIn/mResizeRatio);
}

void px2Cam::CoordTrans_ResizeAndCrop2Ori(int xIn, int yIn, int &xOut, int &yOut)
{
    xOut = (int)((xIn + mROIx)/mResizeRatio);
    yOut = (int)((yIn + mROIy)/mResizeRatio);
}

void px2Cam::InitGL()
{
    if(!mWindow)
    {
        mWindow = new WindowGLFW(mDispParams.windowTitle.c_str(), mDispParams.windowWidth, mDispParams.windowHeight, !mDispParams.onDisplay);
    }
    mWindow->makeCurrent();
}

bool px2Cam::InitSDK()
{
    dwStatus status;

    dwContextParameters sdkParams = {};

    sdkParams.eglDisplay = mWindow->getEGLDisplay();

    status = dwInitialize(&mContext, DW_VERSION, &sdkParams);

    if(status == DW_SUCCESS)
    {
        cout << "[DW_INIT_STEP_1] Driveworks init success" << endl;
    }
    else
    {
        cout << "[DW_INIT_STEP_1] Driveworks init fail" << endl;
        return false;
    }

    return true;
}

bool px2Cam::InitRenderer()
{
    dwStatus status;

    status = dwRenderer_initialize(&mRenderer, mContext);

    if(status == DW_SUCCESS)
    {
        cout << "[DW_INIT_STEP_2] Renderer init success" << endl;
    }
    else
    {
        cout << "[DW_INIT_STEP_2] Renderer init fail" << endl;
        return false;
    }

    dwRenderEngineParams renderEngineParams{};
    CHECK_DW_ERROR(dwRenderEngine_initDefaultParams(&renderEngineParams, mWindow->width(), mWindow->height()));

    renderEngineParams.defaultTile.lineWidth = 0.2f;
    renderEngineParams.defaultTile.font = DW_RENDER_ENGINE_FONT_VERDANA_20;

    CHECK_DW_ERROR(dwRenderEngine_initialize(&mRenderEngine, &renderEngineParams, mContext));

    return true;
}

bool px2Cam::InitSAL()
{
    dwStatus status;

    status = dwSAL_initialize(&mSAL, mContext);

    if(status == DW_SUCCESS)
    {
        cout << "[DW_INIT_STEP_3] SAL init success" << endl;
    }
    else
    {
        cout << "[DW_INIT_STEP_3] SAL init fail" << endl;
        return false;
    }

    return true;
}

bool px2Cam::InitSensors()
{
    dwStatus status;

    dwSensorParams sensorParams;
    memset(&sensorParams, 0, sizeof(dwSensorParams));

    std::string parameterString = std::string("output-format=yuv,fifo-size=3");

    parameterString += std::string(",camera-type=") + mArguments.get("camera-type").c_str();
    parameterString += std::string(",csi-port=") + mArguments.get("csi-port").c_str();
    parameterString += std::string(",slave=") + mArguments.get("slave").c_str();

    if (mArguments.get("custom-board").compare("1") == 0)
    {
        // it's a custom board, use the board specific extra configurations
        parameterString             += ",custom-board=1";

        mArguments.addOption("custom-config");
        mArguments.set("custom-config",
                            "board=E2379a-c01,"
                            "moduleName=ref_max9286_96705_ar0231rccbsf3324ae,"
                            "resolution=1920x1208,"
                            "inputFormat=raw12,"
                            "sensorNum=1,"
                            "interface=csi-ab,"
                            "i2cDevice=7,"
                            "desAddr=0x48,"
                            "brdcstSerAddr=0x40,"
                            "brdcstSensorAddr=0x10");

        sensorParams.auxiliarydata  = mArguments.get("custom-config").c_str();
    }


    sensorParams.parameters = parameterString.c_str();
    sensorParams.protocol = "camera.gmsl";

    status = dwSAL_createSensor(&mCamera, sensorParams, mSAL);

    if(status == DW_SUCCESS)
    {
        cout << "[DW_INIT_STEP_4] Camera init success" << endl;
    }
    else
    {
        cout << "[DW_INIT_STEP_4] Camera init fail : " << dwGetStatusName(status) <<endl;
        return false;
    }

    return true;
}

bool px2Cam::InitPipeline()
{
    dwStatus status;

    status = dwSensor_start(mCamera);

    dwCameraFrameHandle_t frame;
    status = DW_NOT_READY;
    do {
        status = dwSensorCamera_readFrame(&frame, 0, 66000, mCamera);
    } while (status == DW_NOT_READY);

    // something wrong happened, aborting
    if (status != DW_SUCCESS) {
        throw std::runtime_error("Cameras did not start correctly");
    }

    status = dwSensorCamera_returnFrame(&frame);

    status = dwSensorCamera_getSensorProperties(&mCamProp, mCamera);
    printf("Successfully initialized camera with resolution of %dx%d at framerate of %f FPS\n"
           ,mCamProp.resolution.x
           ,mCamProp.resolution.y
           ,mCamProp.framerate);

    // Initialize streamer
    dwImageProperties glImgProps{};
    glImgProps.width = mCamProp.resolution.x;
    glImgProps.height = mCamProp.resolution.y;
    glImgProps.format = DW_IMAGE_FORMAT_RGBA_UINT8;
    glImgProps.type = DW_IMAGE_CUDA;

    status = dwImageStreamer_initialize(&mStreamerCUDA2GL, &glImgProps, DW_IMAGE_GL, mContext);

    if(status == DW_SUCCESS)
    {
        cout << "[DW_INIT_STEP_5] Pipleline init success" << endl;
    }
    else
    {
        cout << "[DW_INIT_STEP_5] Pipleline init fail : " << dwGetStatusName(status) <<endl;
        return false;
    }

    // Init Serializer
    if (mRecordCamera)
    {
        dwSerializerParams seriParams;
        seriParams.parameters = "";
        std::string seriParamsStr = "";
        seriParamsStr += std::string("format=") + std::string(mArguments.get("serializer-type"));
        seriParamsStr += std::string(",bitrate=") + std::string(mArguments.get("serializer-bitrate"));
        seriParamsStr += std::string(",framerate=") + std::string(mArguments.get("serializer-framerate"));
        seriParamsStr += std::string(",type=disk,file=") + std::string(mArguments.get("write-file"));
        seriParamsStr += std::string(",slave=") + std::string(mArguments.get("slave"));

        seriParams.parameters = seriParamsStr.c_str();
        seriParams.onData = nullptr;

        status = dwSensorSerializer_initialize(&mSerializer, &seriParams, mCamera);
        status = dwSensorSerializer_start(mSerializer);

        if(status == DW_SUCCESS)
        {
            cout << "[DW_INIT_STEP_6] Serializer init success" << endl;
        }
        else
        {
            cout << "[DW_INIT_STEP_6] Serializer init fail : " << dwGetStatusName(status) <<endl;
            return false;
        }
    }

    // Allocation Img Data memory
    hipMalloc(&mPitchedImgCudaRGBA, CUDA_PITCH*CAM_IMG_HEIGHT*sizeof(uint8_t));

    hipMalloc(&mGpuMat_data, CAM_IMG_WIDTH*CAM_IMG_HEIGHT*3*sizeof(uint8_t));

    mGpuMat = cv::cuda::GpuMat(CAM_IMG_HEIGHT, CAM_IMG_WIDTH, CV_8UC3, (uint8_t*) mGpuMat_data);

    if(mResizeEnable)
    {
        hipMalloc(&mGpuMatResized_data, mResizeWidth*mResizeHeight*3*sizeof(uint8_t));

        mGpuMatResized = cv::cuda::GpuMat(mResizeHeight, mResizeWidth, CV_8UC3, (uint8_t*)mGpuMatResized_data);
    }

    hipMalloc(&mTrtImg, mROIw*mROIh*3*sizeof(float));

    hipMalloc(&mGpuMatResizedAndCropped_data, mROIw*mROIh*3*sizeof(uint8_t));

    mGpuMatResizedAndCropped = cv::cuda::GpuMat(mROIh, mROIw, CV_8UC3, (uint8_t*)mGpuMatResizedAndCropped_data);

    mMatResizedAndCropped = cv::Mat(mROIh, mROIw, CV_8UC3);

    mMatOri = cv::Mat(CAM_IMG_HEIGHT, CAM_IMG_WIDTH, CV_8UC3);

    return true;
}

bool px2Cam::UpdateCamImg()
{
    dwStatus status;

    status = dwSensorCamera_readFrame(&mFrameHandle, sibling, timeout_us, mCamera);

    if (status == DW_END_OF_STREAM)
    {
        cout << "Camera reached end of stream." << endl;
        return false;
    }
    else if((status == DW_NOT_READY) || (status == DW_TIME_OUT)){
        while((status == DW_NOT_READY) || (status == DW_TIME_OUT))
        {
            std::this_thread::sleep_for(std::chrono::milliseconds(1));
            status = dwSensorCamera_readFrame(&mFrameHandle, sibling, timeout_us, mCamera);
            printf("."); fflush(stdout);
        }
    }
    else if(status == DW_SUCCESS)
    {
//        cout << "[DW_PROC_STEP_1] Read frame success" << endl;
    }
    else
    {
        cout << "[DW_PROC_STEP_1] Read frame fail : " <<  dwGetStatusName(status) << endl;
    }

//    auto begin = std::chrono::high_resolution_clock::now();

    status = dwSensorCamera_getImage(&mFrameCUDAHandle, DW_CAMERA_OUTPUT_CUDA_RGBA_UINT8, mFrameHandle);

    if(status == DW_SUCCESS)
    {
//        cout << "[DW_PROC_STEP_2] Get CUDA frame handle success" << endl;
    }
    else
    {
        cout << "[DW_PROC_STEP_2] Get CUDA frame handle fail : " <<  dwGetStatusName(status) << endl;
    }

    if(mRecordCamera)
    {
        status = dwSensorSerializer_serializeCameraFrameAsync(mFrameHandle, mSerializer);

        if(status == DW_SUCCESS)
        {
//            cout << "[DW_PROC_STEP_2.5] Serializing success" << endl;
        }
        else
        {
            cout << "[DW_PROC_STEP_2.5] Serializing fail : " <<  dwGetStatusName(status) << endl;
        }

    }

    status = dwImage_getCUDA(&mCamImgCuda, mFrameCUDAHandle);

    if(status == DW_SUCCESS)
    {
//        cout << "[DW_PROC_STEP_3] Get CUDA frame success" << endl;
    }
    else
    {
        cout << "[DW_PROC_STEP_3] Get CUDA frame fail : " <<  dwGetStatusName(status) << endl;
    }

    // Get Camera image capture time
    mCamTimestamp = mCamImgCuda->timestamp_us;

    // Copy dwImageCUDA to Pitched pointer
    hipMemcpy(mPitchedImgCudaRGBA, mCamImgCuda->dptr[0], (CUDA_PITCH*CAM_IMG_HEIGHT), hipMemcpyDeviceToDevice);

    const dim3 block(16,16);
    const dim3 grid((CAM_IMG_WIDTH*3 + block.x - 1)/block.x, (CAM_IMG_HEIGHT + block.y -1)/block.y);

    PitchedRGBA2GpuMat <<< grid, block >>> (mPitchedImgCudaRGBA, mGpuMat_data, CAM_IMG_WIDTH, CAM_IMG_HEIGHT, CUDA_PITCH);

    const dim3 gridROI((mROIw*3 + block.x - 1)/block.x, (mROIh + block.y - 1)/block.y);

    if(mResizeEnable)
    {
        cv::cuda::resize(mGpuMat, mGpuMatResized, cv::Size(mResizeWidth, mResizeHeight));

        GpuMat2Img <<< gridROI, block >>> (mGpuMatResized_data, mTrtImg, mGpuMatResizedAndCropped_data,
                                           mResizeWidth, mResizeHeight,
                                           mROIx, mROIy, mROIw, mROIh);
    }
    else
    {
        GpuMat2Img <<< gridROI, block >>> (mGpuMat_data, mTrtImg, mGpuMatResizedAndCropped_data,
                                           CAM_IMG_WIDTH, CAM_IMG_HEIGHT,
                                           mROIx, mROIy, mROIw, mROIh);
    }

    dwSensorCamera_returnFrame(&mFrameHandle);

//    auto end = std::chrono::high_resolution_clock::now();

//    cout << chrono::duration_cast<chrono::milliseconds>(end - begin).count() << "ms" << endl;

    return true;
}

void px2Cam::RenderCamImg()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    dwTime_t timeout = 132000;

    // stream that image to the GL domain
    CHECK_DW_ERROR(dwImageStreamer_producerSend(mFrameCUDAHandle, mStreamerCUDA2GL));

    CHECK_DW_ERROR(dwImageStreamer_consumerReceive(&mFrameGLHandle, timeout, mStreamerCUDA2GL));

    CHECK_DW_ERROR(dwImage_getGL(&mImgGl, mFrameGLHandle));

    // render received texture
    dwVector2f range{};
    range.x = mImgGl->prop.width;
    range.y = mImgGl->prop.height;
    CHECK_DW_ERROR(dwRenderEngine_setCoordinateRange2D(range, mRenderEngine));
    CHECK_DW_ERROR(dwRenderEngine_renderImage2D(mImgGl, {0.0f, 0.0f, range.x, range.y}, mRenderEngine));

    // returned the consumed image
    CHECK_DW_ERROR(dwImageStreamer_consumerReturn(&mFrameGLHandle, mStreamerCUDA2GL));

    // notify the producer that the work is done
    CHECK_DW_ERROR(dwImageStreamer_producerReturn(nullptr, timeout, mStreamerCUDA2GL));
}

void px2Cam::DrawBoundingBoxes(vector<cv::Rect>  bbRectList, vector<float32_t*> bbColorList, float32_t lineWidth)
{
    CHECK_DW_ERROR(dwRenderEngine_setLineWidth(lineWidth, mRenderEngine));
    for(uint bbInd = 0; bbInd < bbRectList.size(); bbInd++)
    {
        float32_t* bBoxColor = bbColorList[bbInd];
        dwRenderEngineColorRGBA bBoxColorDw;
        bBoxColorDw.x = bBoxColor[0];
        bBoxColorDw.y = bBoxColor[1];
        bBoxColorDw.z = bBoxColor[2];
        bBoxColorDw.w = bBoxColor[3];
        CHECK_DW_ERROR(dwRenderEngine_setColor(bBoxColorDw, mRenderEngine));

        cv::Rect bBoxRect = bbRectList[bbInd];
        dwRectf bBoxRectDw;
        bBoxRectDw.x = bBoxRect.x;
        bBoxRectDw.y = bBoxRect.y;
        bBoxRectDw.width = bBoxRect.width;
        bBoxRectDw.height = bBoxRect.height;

        dwRenderEngine_render(DW_RENDER_ENGINE_PRIMITIVE_TYPE_BOXES_2D, &bBoxRectDw, sizeof(dwRectf), 0, 1, mRenderEngine);
    }
}

void px2Cam::DrawBoundingBoxesWithLabels(vector<cv::Rect>  bbRectList, vector<float32_t*> bbColorList, vector<const char*> bbLabelList, float32_t lineWidth)
{
    CHECK_DW_ERROR(dwRenderEngine_setLineWidth(lineWidth, mRenderEngine));
    for(uint bbInd = 0; bbInd < bbRectList.size(); bbInd++)
    {
        float32_t* bBoxColor = bbColorList[bbInd];
        dwRenderEngineColorRGBA bBoxColorDw;
        bBoxColorDw.x = bBoxColor[0];
        bBoxColorDw.y = bBoxColor[1];
        bBoxColorDw.z = bBoxColor[2];
        bBoxColorDw.w = bBoxColor[3];
        CHECK_DW_ERROR(dwRenderEngine_setColor(bBoxColorDw, mRenderEngine));

        cv::Rect bBoxRect = bbRectList[bbInd];
        dwRectf bBoxRectDw;
        bBoxRectDw.x = bBoxRect.x;
        bBoxRectDw.y = bBoxRect.y;
        bBoxRectDw.width = bBoxRect.width;
        bBoxRectDw.height = bBoxRect.height;

        const char* bbLabel = bbLabelList[bbInd];

        dwRenderEngine_renderWithLabel(DW_RENDER_ENGINE_PRIMITIVE_TYPE_BOXES_2D, &bBoxRectDw, sizeof(dwRectf), 0, bbLabel, 1, mRenderEngine);
    }
}

void px2Cam::DrawBoundingBoxesWithLabelsPerClass(vector<vector<dwRectf> >  bbRectList, vector<const float32_t*> bbColorList, vector<vector<const char*> > bbLabelList, float32_t lineWidth)
{
    CHECK_DW_ERROR(dwRenderEngine_setLineWidth(lineWidth, mRenderEngine));
    for(uint classIdx = 0; classIdx < bbRectList.size(); classIdx++)
    {
        const float32_t* bBoxColor = bbColorList[classIdx];
        dwRenderEngineColorRGBA bBoxColorDw;
        bBoxColorDw.x = bBoxColor[0];
        bBoxColorDw.y = bBoxColor[1];
        bBoxColorDw.z = bBoxColor[2];
        bBoxColorDw.w = bBoxColor[3];
        CHECK_DW_ERROR(dwRenderEngine_setColor(bBoxColorDw, mRenderEngine));

        if (bbRectList[classIdx].size() == 0)
            continue;

        CHECK_DW_ERROR(dwRenderEngine_renderWithLabels(DW_RENDER_ENGINE_PRIMITIVE_TYPE_BOXES_2D, &bbRectList[classIdx][0], sizeof(dwRectf), 0, &bbLabelList[classIdx][0], bbRectList[classIdx].size(), mRenderEngine));
    }
}

void px2Cam::DrawPoints(vector<cv::Point> ptList, float32_t ptSize, float32_t* ptColor)
{
    CHECK_DW_ERROR(dwRenderEngine_setPointSize(ptSize, mRenderEngine));
    vector<dwVector2f> ptDwList;
    for(uint ptInd = 0; ptInd < ptList.size(); ptInd++)
    {
        cv::Point pt = ptList[ptInd];
        dwVector2f ptDw;
        ptDw.x = pt.x;
        ptDw.y = pt.y;
        ptDwList.push_back(ptDw);
    }

    dwRenderEngineColorRGBA ptColorDw;
    ptColorDw.x = ptColor[0];
    ptColorDw.y = ptColor[1];
    ptColorDw.z = ptColor[2];
    ptColorDw.w = ptColor[3];

    CHECK_DW_ERROR(dwRenderEngine_setColor(ptColorDw, mRenderEngine));

    dwRenderEngine_render(DW_RENDER_ENGINE_PRIMITIVE_TYPE_POINTS_2D, &ptDwList[0], sizeof(dwVector2f), 0, ptDwList.size(), mRenderEngine);
}

void px2Cam::DrawPolyLine(vector<cv::Point> ptList, float32_t lineWidth, float32_t* lineColor)
{
    CHECK_DW_ERROR(dwRenderEngine_setLineWidth(lineWidth, mRenderEngine));
    vector<dwVector2f> ptDwList;
    for(uint ptInd = 0; ptInd < ptList.size(); ptInd++)
    {
        cv::Point pt = ptList[ptInd];
        dwVector2f ptDw;
        ptDw.x = pt.x;
        ptDw.y = pt.y;
        ptDwList.push_back(ptDw);
    }

    dwRenderEngineColorRGBA lineColorDw;
    lineColorDw.x = lineColor[0];
    lineColorDw.y = lineColor[1];
    lineColorDw.z = lineColor[2];
    lineColorDw.w = lineColor[3];

    CHECK_DW_ERROR(dwRenderEngine_setColor(lineColorDw, mRenderEngine));

    dwRenderEngine_render(DW_RENDER_ENGINE_PRIMITIVE_TYPE_LINESTRIP_2D, &ptDwList[0], sizeof(dwVector2f), 0, ptDwList.size(), mRenderEngine);
}

void px2Cam::DrawPolyLineDw(vector<dwVector2f> ptList, float32_t lineWidth, dwVector4f lineColor)
{
    CHECK_DW_ERROR(dwRenderEngine_setLineWidth(lineWidth, mRenderEngine));

    CHECK_DW_ERROR(dwRenderEngine_setColor(lineColor, mRenderEngine));

    dwRenderEngine_render(DW_RENDER_ENGINE_PRIMITIVE_TYPE_LINESTRIP_2D, &ptList[0], sizeof(dwVector2f), 0, ptList.size(), mRenderEngine);
}

void px2Cam::DrawText(const char* text, cv::Point textPos, float32_t* textColor)
{
    dwVector2f textPosDw;
    textPosDw.x = textPos.x;
    textPosDw.y = textPos.y;


    dwRenderEngineColorRGBA textColorDw;
    textColorDw.x = textColor[0];
    textColorDw.y = textColor[1];
    textColorDw.z = textColor[2];
    textColorDw.w = textColor[3];

    CHECK_DW_ERROR(dwRenderEngine_setColor(textColorDw, mRenderEngine));

    dwRenderEngine_renderText2D(text, textPosDw, mRenderEngine);
}

void px2Cam::UpdateRendering()
{
    mWindow->swapBuffers();
}

dwContextHandle_t px2Cam::GetDwContext()
{
    return mContext;
}

trtImgData px2Cam::GetTrtImgData()
{
    mCurTrtImgData.timestamp_us = mCamTimestamp;
    mCurTrtImgData.trtImg = mTrtImg;
    return mCurTrtImgData;
}

matImgData px2Cam::GetCroppedMatImgData()
{
    mGpuMatResizedAndCropped.download(mMatResizedAndCropped);
    mCurCroppedMatImgData.timestamp_us = mCamTimestamp;
    mCurCroppedMatImgData.matImg = mMatResizedAndCropped;
    return mCurCroppedMatImgData;
}

matImgData px2Cam::GetOriMatImgData()
{
    mGpuMat.download(mMatOri);
    mCurOriMatImgData.timestamp_us = mCamTimestamp;
    mCurOriMatImgData.matImg = mMatOri;
    return mCurOriMatImgData;
}

dwImageCUDA* px2Cam::GetDwImageCuda()
{
    return mCamImgCuda;
}
